
#include <hip/hip_runtime.h>
#include  <iostream>
#include  <math.h>
 
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
      y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20; // 1M elements

  // float *x = new float[N];
  // float *y = new float[N];
  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
  add<<<1, 1>>>(N, x, y);

  // wait for cuda to finish before continuing
  hipDeviceSynchronize();


  float maxError = 0.0f;
  for (int i = 0; i < N; i++) {
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  }
  std::cout << "Max error: " << maxError << std::endl;

  hipFree(x);
  hipFree(y);
  // delete [] x;
  // delete [] y;

  return 0;
}

